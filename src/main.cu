
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <stdio.h>

class CudaClass {
public:
  int x = 4;
  double* v;
  CudaClass(int size) {
    v = new double[size];
    for(int i = 0; i < size; ++i)
        v[i] = i;
  }
  ~CudaClass() {
    delete[] v;
  }
};

__global__ void func(CudaClass* c) {
    printf("x: %d\n", c->x);
    //for(int i = 0; i < size; ++i)
        //std::cout<<c->v[i]<<std::endl;
}


int main() {
  CudaClass c(8);
  CudaClass* d_c;
  hipMalloc((void **)&d_c, sizeof(CudaClass));
  hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice);
  func<<<1,1>>>(d_c);
}

